#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include <string.h>
#include <time.h>
#include <hip/hip_runtime_api.h>
#include "sort.cu"
#include "cuda_commons.cu"
void check_error(hipError_t err, const char *msg);
void knnParallel(float* coords, float* newCoords, int* classes, int numClasses, int numSamples, int numNewSamples, int k);

__device__ float manhattan_distance_gpu(float x, float y) {
    return fabsf(x - y);
}



__global__ void distances_kernel_naive(float* dataset, float* to_predict, int dataset_n, int dimension,
                           int to_predict_n, float* distances, int distance_algorithm) {
    // Cada hilo en x, y guarda la distancia entre el vector x del dataset y el vector
    // y a predecir
    // distances tiene filas de to_predict_n de ancho
    // cada fila tiene todas las distancias para el to_pred_i contra todos los del dastaset
    int dataset_i = blockIdx.x * blockDim.x + threadIdx.x;
    int to_pred_i = blockIdx.y * blockDim.y + threadIdx.y;

    if (dataset_i >= dataset_n || to_pred_i >= to_predict_n)
        return;

    float distance = 0;

    for (int i = 0; i < dimension ; i++) {
            distance += manhattan_distance_gpu(
                to_predict[to_pred_i * dimension + i],
                dataset[dataset_i * dimension + i]
            );
    }
    distances[to_pred_i * dataset_n + dataset_i] = distance;
}
void knnParallel(float* coords, float* newCoords, int* classes, int numClasses, int numSamples, int numNewSamples, int k) {
    //*** Device-variables-declaration ***
    float* d_coords;
    float* d_newCoords;
    int* d_classes;
    float *distance;
    int totalSamples = numSamples + numNewSamples;
    int *gpu_tags, *gpu_results, *gpu_winners;
    //*** device-allocation ***
    check_error(hipMalloc(&d_coords, totalSamples * DIMENSION * sizeof(float)), "alloc d_coords_x");
    check_error(hipMalloc(&d_classes, totalSamples * sizeof(int)), "alloc d_classes");
    check_error(hipMalloc(&d_newCoords, numNewSamples * DIMENSION * sizeof(float)), "alloc d_coordsnew");
    CUDA_CHK(hipMalloc((void**)&distance, numSamples * numNewSamples * sizeof(float)));
    //***copy-arrays-on-device***
    check_error(hipMemcpy(d_coords, coords, totalSamples * DIMENSION * sizeof(float), hipMemcpyHostToDevice), "copy d_coords");
    check_error(hipMemcpy(d_classes, classes, totalSamples * sizeof(int), hipMemcpyHostToDevice), "copy d_classes");
    check_error(hipMemcpy(d_newCoords, newCoords, numNewSamples * DIMENSION * sizeof(float), hipMemcpyHostToDevice), "copy d_coordsnew");
    CUDA_CHK(hipMalloc((void**)&gpu_results, k * numNewSamples * sizeof(int)));
    CUDA_CHK(hipMalloc((void**)&gpu_winners, totalSamples * sizeof(int)));
    // TODO: Put your parallel code in this function
    /*
       1. Design the KNN parallel code.
       1. Specify the sizes of grid and block.
       2. Launch the kernel function (Write kernel code in knnCUDA.cu).
    */
    dim3 tamGrid, tamBlock;
    int block_size=32;
    tamGrid = dim3(numSamples / block_size, numNewSamples / block_size);
    tamBlock = dim3(block_size, block_size);
    if (numSamples % block_size != 0) tamGrid.x += 1;
    if (numNewSamples % block_size != 0) tamGrid.y += 1;

    distances_kernel_naive <<< tamGrid, tamBlock >>> (
            coords, newCoords, numSamples, 2, numNewSamples, distance
        );
    quick_sort(classes, numSamples, numNewSamples, k, &distance, gpu_results);
    int *results = (int*)malloc(k * numNewSamples * sizeof(int));
    CUDA_CHK(hipMemcpy(results, gpu_results, k * numNewSamples * sizeof(int), hipMemcpyDeviceToHost));
    int count_grid_width = k < numSamples ? numSamples : k;
    dim3 tamGrid_count(count_grid_width / block_size, numNewSamples / block_size);
    if (count_grid_width % block_size != 0) tamGrid_count.x += 1;
    if (numNewSamples % block_size != 0) tamGrid_count.y += 1;
    dim3 tamBlock_count(block_size, block_size);
    int shared_size = (k * numNewSamples + numSamples) * sizeof(int);
    count_winner_kernel <<< tamGrid_count, tamBlock_count, shared_size >>> (gpu_results, gpu_winners, numNewSamples, k, numSamples);
    check_error(hipMemcpy(d_classes, gpu_winners, numNewSamples * sizeof(int), hipMemcpyDeviceToHost));
    hipDeviceSynchronize();
    // download device -> host
    check_error(hipMemcpy(coords, d_coords, DIMENSION * totalSamples * sizeof(float), hipMemcpyDeviceToHost), "download coords");
    check_error(hipMemcpy(classes, d_classes, totalSamples * sizeof(int), hipMemcpyDeviceToHost), "download classes");
   
}

void check_error(hipError_t err, const char *msg)
{
    if (err != hipSuccess) 
    {
        fprintf(stderr, "%s : error %d (%s)\n", msg, err, hipGetErrorString(err));
        exit(err);
    }
}
